#include "hip/hip_runtime.h"
#include<stdint.h>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdio.h>

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;

void timing(real* h_x, real* d_x, const int method);

int main(void)
{
    real* h_x = (real*)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real* d_x;
    CHECK(hipMalloc(&d_x, M));

    printf("\nUsing global memory only:\n");
    timing(h_x, d_x, 0);
    printf("\nUsing static shared memory:\n");
    timing(h_x, d_x, 1);
    printf("\nUsing dynamic shared memory:\n");
    timing(h_x, d_x, 2);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ reduce_global(real* d_x, real* d_y)
{
    const int tid = threadIdx.x;
 //����ָ��X���ұ߱�ʾ d_x �����  blockDim.x * blockIdx.x��Ԫ�صĵ�ַ
 //�������x �ڲ�ͬ�߳̿飬ָ��ȫ���ڴ治ͬ�ĵ�ַ---��ʹ�ò�ͬ���߳̿��dx���鲻ͬ���ֱַ���д���   
    real* x = d_x + blockDim.x * blockIdx.x;

    //blockDim.x >> 1  �ȼ��� blockDim.x /2���˺����У�λ������ ��Ӧ������������Ч
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            x[tid] += x[tid + offset];
        }
        //ͬ����䣬���ã�ͬһ���߳̿��ڵ��̰߳��մ����Ⱥ�ִ��ָ�����ͬ�������ⲻ��ͬ����
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[blockIdx.x] = x[0];
    }
}

void __global__ reduce_shared(real* d_x, real* d_y)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    //�����˹����ڴ����� s_y[128]��ע��ؼ���  __shared__
    __shared__ real s_y[128];
    //��ȫ���ڴ��е����ݸ��Ƶ������ڴ���
    //�����ڴ���� ����ÿ���߳̿鶼��һ�������ڴ�����ĸ���
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    //���ú��� __syncthreads �����߳̿��ڵ�ͬ��
    __syncthreads();
    //��Լ�����ù����ڴ�����滻��ԭ����ȫ���ڴ����������ҲҪ��ס�� ÿ���߳̿鶼�����еĹ����ڴ�����������в������ڹ�Լ���̽�����ÿһ���߳�
    //���е� s_y[0] �����ͱ�������������Ԫ�صĺ�
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_y[bid] = s_y[0];
    }
}

void __global__ reduce_dynamic(real* d_x, real* d_y)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    //���� ��̬�����ڴ� s_y[]  �޶��� extern������ָ�������С
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {

        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {//��ÿһ���߳̿��й�Լ�Ľ���ӹ����ڴ� s_y[0] ���Ƶ�ȫ���� ��d_y[bid]
        d_y[bid] = s_y[0];
    }
}

real reduce(real* d_x, const int method)
{
    int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int ymem = sizeof(real) * grid_size;
    const int smem = sizeof(real) * BLOCK_SIZE;
    real* d_y;
    CHECK(hipMalloc(&d_y, ymem));
    real* h_y = (real*)malloc(ymem);

    switch (method)
    {
    case 0:
        reduce_global << <grid_size, BLOCK_SIZE >> > (d_x, d_y);
        break;
    case 1:
        reduce_shared << <grid_size, BLOCK_SIZE >> > (d_x, d_y);
        break;
    case 2:
        reduce_dynamic << <grid_size, BLOCK_SIZE, smem >> > (d_x, d_y);
        break;
    default:
        printf("Error: wrong method\n");
        exit(1);
        break;
    }

    CHECK(hipMemcpy(h_y, d_y, ymem, hipMemcpyDeviceToHost));

    real result = 0.0;
    for (int n = 0; n < grid_size; ++n)
    {
        result += h_y[n];
    }

    free(h_y);
    CHECK(hipFree(d_y));
    return result;
}

void timing(real* h_x, real* d_x, const int method)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(d_x, method);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}


