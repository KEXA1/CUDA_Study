#include<stdint.h>
#include<hip/hip_runtime.h>


#include <stdio.h>
#include <math.h>
#include <stdio.h>

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


//�ж�ʹ�� ������ or˫���� ������
#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 20;
// ��ʱ
void timing(const real* x, const int N);
//����Ԫ����� NΪԪ�ظ��� xΪ�����ַ
real reduce(const real* x, const int N);

int main(void)
{// 1�ڸ�Ԫ��
    const int N = 100000000;
    //���ٿռ䣬��СΪ M bytes
    const int M = sizeof(real) * N;
    real* x = (real*)malloc(M);
    //��ʼ������
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.23;
    }

    timing(x, N);

    free(x);
    return 0;
}

void timing(const real* x, const int N)
{
    real sum = 0;

    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        sum = reduce(x, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    printf("sum = %f.\n", sum);
}

real reduce(const real* x, const int N)
{
    real sum = 0.0;
    for (int n = 0; n < N; ++n)
    {
        sum += x[n];
    }
    return sum;
}


