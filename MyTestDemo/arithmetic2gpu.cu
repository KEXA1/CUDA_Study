#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include<stdint.h>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)


#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 10;
const real x0 = 100.0;
void __global__ arithmetic(real* x, const real x0, const int N);

int main(void)
{
    const int N = 10000;
   // const int N = atoi(argv[1]);
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    const int M = sizeof(real) * N;
    real* h_x = (real*)malloc(M);
    real* d_x;
    CHECK(hipMalloc((void**)&d_x, M));

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            h_x[n] = 0.0;
        }
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        arithmetic << <grid_size, block_size >> > (d_x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ arithmetic(real* d_x, const real x0, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        real x_tmp = d_x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        d_x[n] = x_tmp;
    }
}


