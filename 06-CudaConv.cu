#include "hip/hip_runtime.h"
#include<stdint.h>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

const int NUM_REPEATS = 10;

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)



static void HandleError(hipError_t err,
    const char* file,

    int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n",
            hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;

    CHECK(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    CHECK(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
        prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

__global__ void conv(float* img, float* kernel, float* result,
    int width, int height, int kernelSize)
{
    int ti = threadIdx.x;
    int bi = blockIdx.x;
    int id = (bi * blockDim.x + ti);
    if (id >= width * height)
    {
        return;
    }
    int row = id / width;
    int col = id % width;
    for (int i = 0; i < kernelSize; ++i)
    {
        for (int j = 0; j < kernelSize; ++j)
        {
            float imgValue = 0;
            int curRow = row - kernelSize / 2 + i;
            int curCol = col - kernelSize / 2 + j;
            if (curRow < 0 || curCol < 0 || curRow >= height || curCol >= width)
            {
            }
            else
            {
                imgValue = img[curRow * width + curCol];
            }
            result[id] += kernel[i * kernelSize + j] * imgValue;
        }

    }
}

int main()
{
    int width = 1000;
    int height = 1000;
    float* img = new float[width * height];
    for (int row = 0; row < height; ++row)
    {
        for (int col = 0; col < width; ++col)
        {
            img[col + row * width] = (col + row) % 256;
        }
    }

    int kernelSize = 3;
    float* kernel = new float[kernelSize * kernelSize];
    for (int i = 0; i < kernelSize * kernelSize; ++i)
    {
        kernel[i] = i % kernelSize - 1;
    }

    float* imgGpu;
    float* kernelGpu;
    float* resultGpu;

    CHECK(hipMalloc((void**)&imgGpu, width * height * sizeof(float)));
    CHECK(hipMalloc((void**)&kernelGpu, kernelSize * kernelSize * sizeof(float)));
    CHECK(hipMalloc((void**)&resultGpu, width * height * sizeof(float)));

    CHECK(hipMemcpy(imgGpu, img,
        width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(kernelGpu, kernel,
        kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice));

    int threadNum = getThreadNum();
    int blockNum = (width * height - 0.5) / threadNum + 1;

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        conv << <blockNum, threadNum >> >
            (imgGpu, kernelGpu, resultGpu, width, height, kernelSize);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);


    float* result = new float[width * height];
    CHECK(hipMemcpy(result, resultGpu,
        width * height * sizeof(float), hipMemcpyDeviceToHost));

    // visualization
    printf("img\n");
    for (int row = 0; row < 10; ++row)
    {
        for (int col = 0; col < 10; ++col)
        {
            printf("%2.0f ", img[col + row * width]);
        }
        printf("\n");
    }
    printf("kernel\n");
    for (int row = 0; row < kernelSize; ++row)
    {
        for (int col = 0; col < kernelSize; ++col)
        {
            printf("%2.0f ", kernel[col + row * kernelSize]);
        }
        printf("\n");
    }

    printf("result\n");
    for (int row = 0; row < 10; ++row)
    {
        for (int col = 0; col < 10; ++col)
        {
            printf("%2.0f ", result[col + row * width]);
        }
        printf("\n");
    }


    return 0;
}
