#include <math.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include<stdint.h>
#include<hip/hip_runtime.h>


#include <stdio.h>

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

const int NUM_REPEATS = 10;
const real x0 = 100.0;
void arithmetic(real* x, const real x0, const int N);

int main(void)
{
    const int N = 10000;
    const int M = sizeof(real) * N;
    real* x = (real*)malloc(M);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        for (int n = 0; n < N; ++n)
        {
            x[n] = 0.0;
        }

        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        arithmetic(x, x0, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    free(x);
    return 0;
}

void arithmetic(real* x, const real x0, const int N)
{
    for (int n = 0; n < N; ++n)
    {
        real x_tmp = x[n];
        while (sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        x[n] = x_tmp;
    }
}
